#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace cv;
using namespace std;

//GLOBALS
int thresholdSlider;
const int THRESHOLD_SLIDER_MAX = 255;
hipDeviceProp_t deviceProps;
Mat hostImage;
unsigned char THRESHOLD = 120;
unsigned char *dev0_image;
unsigned char *devCopy_image;
int imageSize = 0;
//GLOBALS

__global__ void kernel(unsigned char* imageOrig, unsigned char* imageCopy, unsigned char threshold)
{

	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (imageOrig[j] > threshold)
	{
		imageCopy[j] = 255;
	}
	else
	{
		imageCopy[j] = 0;
	}
}

void Threshold(Mat image, unsigned char threshold);
void thresholdWithCuda(Mat* image, unsigned char threshold);

void on_Trackbar(int, void *)
{
	int blocksNeeded = (imageSize + deviceProps.maxThreadsPerBlock - 1) / deviceProps.maxThreadsPerBlock;
	//use kernel to threshold dev0_image, then write to devCopy_image
	kernel << <blocksNeeded, deviceProps.maxThreadsPerBlock >> >(dev0_image, devCopy_image, thresholdSlider);
	hipDeviceSynchronize();

	if (hipMemcpy(hostImage.data, devCopy_image, imageSize, hipMemcpyDeviceToHost) != hipSuccess)
	{
		throw("trackbar memcopy failed");
	}

	imshow("Display window", hostImage);
}

int main(int argc, char** argv)
{

	//if 2 arguements arent passed, tell user there was an error
	if (argc != 2) {
		cout << "Usage: display_image ImageToLoadAndDisplay" << endl;
		return -1;
	}

	//set hostImage to command argument
	hostImage = imread(argv[1], CV_LOAD_IMAGE_COLOR);

	if (!hostImage.data) {
		cout << "Could not open or find the image" << endl;
		return -1;
	}

	//display image information
	cout << "Image has: " << hostImage.channels() << " channels" << endl;
	cout << "Image is " << hostImage.cols << "x" << hostImage.rows << endl;

	//convert image to grayscale
	cvtColor(hostImage, hostImage, cv::COLOR_RGB2GRAY);

	thresholdWithCuda(&hostImage, THRESHOLD);

	namedWindow("Display window", WINDOW_NORMAL);
	resizeWindow("Display window", 1900, 1080);
	createTrackbar("Threshold", "Display window", &thresholdSlider, THRESHOLD_SLIDER_MAX, on_Trackbar);
	imshow("Display window", hostImage);

	waitKey(0);
	return 0;
}

void Threshold(Mat hostImage, unsigned char threshold)
{
	int height = hostImage.rows;
	int width = hostImage.cols;

	for (int i = 0; i < height*width; i++)
	{
		if (hostImage.data[i] > threshold)
		{
			hostImage.data[i] = 255;
		}
		else
		{
			hostImage.data[i] = 0;
		}
	}
}
void thresholdWithCuda(Mat* hostImage, unsigned char threshold)
{
	hipError_t cudaStatus;

	int height = hostImage->rows;
	int width = hostImage->cols;
	imageSize = height * width;

	try
	{
		//set device
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
		{
			throw("error in set device");
		}
		//check device props
		cudaStatus = hipGetDeviceProperties(&deviceProps, 0);
		if (cudaStatus != hipSuccess) {
			throw("getDeviceProperties failed");
		}
		//malloc original image and copy image
		cudaStatus = hipMalloc((void**)&dev0_image, imageSize);
		if (cudaStatus != hipSuccess)
		{
			throw("hipMalloc failed");
		}

		cudaStatus = hipMalloc((void**)&devCopy_image, imageSize);
		if (cudaStatus != hipSuccess)
		{
			throw("hipMalloc failed");
		}

		//copy original to gpu
		cudaStatus = hipMemcpy(dev0_image, hostImage->data, imageSize, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			throw("mem copy failed");
		}

		int blocksNeeded = (imageSize + deviceProps.maxThreadsPerBlock - 1) / deviceProps.maxThreadsPerBlock;
		//use kernel to threshold dev0_image, then write to devCopy_image
		kernel << <blocksNeeded, deviceProps.maxThreadsPerBlock >> >(dev0_image, devCopy_image, threshold);
		if (hipGetLastError() != hipSuccess)
			throw("add Kernel failed");
		cudaStatus = hipDeviceSynchronize();

		cudaStatus = hipMemcpy(hostImage->data, devCopy_image, imageSize, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			throw("mem copy failed");
		}
	}
	catch (char* error)
	{
		cout << error << endl;
		goto bad_exit;
	}
bad_exit:
	hipFree((void*)&dev0_image);
	hipFree((void*)&devCopy_image);
}